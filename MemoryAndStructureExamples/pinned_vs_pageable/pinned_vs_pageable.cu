#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <vector>

// Move this out to top level.
#ifndef cudaCheckError
#define cudaCheckError(call)                                                                                           \
    {                                                                                                                  \
        auto status = static_cast<hipError_t>( call );                                                                \
        if ( status != hipSuccess )                                                                                   \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
                     "with "                                                                                           \
                     "%s (%d).\n",                                                                                     \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     hipGetErrorString( status ),                                                                     \
                     status );                                                                                         \
    }
#endif  // cudaCheckError


__global__ void vector_add_kernel(float *d_v1, float *d_v2, float *d_v3, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        d_v3[i] = d_v1[i] + d_v2[i];
    }
}

// Structure to pass around host memory pointers.
struct Memory {
    float *v1;
    float *v2;
    float *v3;
};

void benchmark(int N, bool usePinnedMemory) {
    size_t size = N * sizeof(float);
    Memory mem;
    
    // Allocate memory
    if (usePinnedMemory) {
        cudaCheckError(hipHostMalloc(&mem.v1, size, hipHostMallocDefault));
        cudaCheckError(hipHostMalloc(&mem.v2, size, hipHostMallocDefault));
        cudaCheckError(hipHostMalloc(&mem.v3, size, hipHostMallocDefault));
    } else {
        mem.v1 = (float*)malloc(size);
        mem.v2 = (float*)malloc(size);
        mem.v3 = (float*)malloc(size);
    }

    // Initialize data
    for (int i = 0; i < N; i++) {
        mem.v1[i] = 1.0f;
        mem.v2[i] = 2.0f;
    }

    // Allocate device memory
    float *d_v1, *d_v2, *d_v3;
    cudaCheckError(hipMalloc(&d_v1, size));
    cudaCheckError(hipMalloc(&d_v2, size));
    cudaCheckError(hipMalloc(&d_v3, size));

    // Create CUDA stream
    hipStream_t stream;
    cudaCheckError(hipStreamCreate(&stream));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    cudaCheckError(hipEventCreate(&start));
    cudaCheckError(hipEventCreate(&stop));

    // Record start time
    cudaCheckError(hipEventRecord(start, stream));

    // Asynchronous memory transfers
    cudaCheckError(hipMemcpyAsync(d_v1, mem.v1, size, hipMemcpyHostToDevice, stream));
    cudaCheckError(hipMemcpyAsync(d_v2, mem.v2, size, hipMemcpyHostToDevice, stream));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_v1, d_v2, d_v3, N);

    // Asynchronous memory transfer back to host
    cudaCheckError(hipMemcpyAsync(mem.v3, d_v3, size, hipMemcpyDeviceToHost, stream));

    // Record stop time
    cudaCheckError(hipEventRecord(stop, stream));
    cudaCheckError(hipEventSynchronize(stop));

    float milliseconds = 0;
    cudaCheckError(hipEventElapsedTime(&milliseconds, start, stop));

    // Calculate bandwidth
    float bandwidth = 3 * size / (milliseconds / 1000) / 1e9; // GB/s

    std::cout << (usePinnedMemory ? "Pinned" : "Pageable") << " memory, N = " << N 
              << ", Time: " << milliseconds << " ms, Bandwidth: " << bandwidth << " GB/s" << std::endl;

    // Clean up
    cudaCheckError(hipFree(d_v1));
    cudaCheckError(hipFree(d_v2));
    cudaCheckError(hipFree(d_v3));
    cudaCheckError(hipStreamDestroy(stream));
    cudaCheckError(hipEventDestroy(start));
    cudaCheckError(hipEventDestroy(stop));

    if (usePinnedMemory) {
        cudaCheckError(hipHostFree(mem.v1));
        cudaCheckError(hipHostFree(mem.v2));
        cudaCheckError(hipHostFree(mem.v3));
    } else {
        free(mem.v1);
        free(mem.v2);
        free(mem.v3);
    }
}

int main() {
    std::vector<int> sizes = {1024, 1024 * 1024, 10 * 1024 * 1024};
    
    for (int N : sizes) {
        benchmark(N, false); // Pageable memory
        benchmark(N, true);  // Pinned memory
        std::cout << std::endl;
    }

    return 0;
}
