#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <nvToolsExt.h>

#define W 1024

__global__ void matMul(float* out, const float* in1, const float* in2, const int width) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < width && col < width) {
        for (int i = 0; i < width; ++i) {
            sum += in1[row * width + i] * in2[i * width + col];
        }
        out[row * width + col] = sum;
    }
}

void doMatMul(float* out, const float* in1, const float* in2, int width) {
    float *dIn1, *dIn2, *dOut;
    const size_t size = width * width * sizeof(float);

    printf("Allocating device memory...\n");
    hipMalloc(&dIn1, size);
    hipMalloc(&dIn2, size);
    hipMalloc(&dOut, size);

    printf("Copying data to device...\n");
    hipMemcpy(dIn1, in1, size, hipMemcpyHostToDevice);
    hipMemcpy(dIn2, in2, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (width + threadsPerBlock.y - 1) / threadsPerBlock.y);
    printf("Launching kernel...\n");
    for (int j = 0; j < 10; ++j)
    {
        nvtxRangePush("Matrix multiplication kernel");
        matMul<<<blocksPerGrid, threadsPerBlock>>>(dOut, dIn1, dIn2, width);
        nvtxRangePop();
    }
    hipDeviceSynchronize();

    printf("Copying result to host...\n");
    hipMemcpy(out, dOut, size, hipMemcpyHostToDevice);

    printf("Freeing device memory...\n");
    hipFree(dIn1);
    hipFree(dIn2);
    hipFree(dOut);
}

int main() {
    float *in1, *in2, *out;
    const size_t width = W * W * sizeof(float);
    in1 = (float*)malloc(width);
    in2 = (float*)malloc(width);
    out = (float*)malloc(width);
    for (int i = 0; i < W * W; ++i) {
        in1[i] = static_cast<float>(rand()) / RAND_MAX;
        in2[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    printf("Starting matrix multiplication...\n");
    doMatMul(out, in1, in2, width);
    printf("Matrix multiplication completed.\n");
    printf("Freeing host memory...\n");
    free(in1);
    free(in2);
    free(out);
    return 0;
}
