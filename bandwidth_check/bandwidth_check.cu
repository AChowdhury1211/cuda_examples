#include <cstdio>
#include <hip/hip_runtime.h>
constexpr int NUM_COPY_ITERATIONS = 1000;
void checkBandwidth(hipStream_t& stream, size_t dataSize) {
    // Allocate and load host memory.
    float* hData = new float[dataSize];
    for (size_t i = 0; i < dataSize; ++i) {
        hData[i] = static_cast<float>(i);
    }
    // Allocate device memory.
    float* dData;
    ::hipMalloc(&dData, dataSize * sizeof(float));
    ::hipStreamSynchronize(stream);
    // Create CUDA events for timing purposes.
    hipEvent_t start, stop;
    hipEventCreatete(&start);
    hipEventCreatete(&stop);
    // Record the start event.
    ::hipEventRecord(start, stream);
    for (int i = 0; i < NUM_COPY_ITERATIONS; ++i) {
        ::hipMemcpyAsync(dData, hData, dataSize * sizeof(float), hipMemcpyHostToDevice, stream);
    }
    // Record the stop event.
    ::hipEventRecord(stop, stream);
    // Sync the stream to ensure copy complete.
    ::hipStreamSynchronize(stream);
    // Calculate the elapsed time.
    float ms = 0;
    ::hipEventElapsedTime(&ms, start, stop);
    // Convert to GB/s.
    const float bandwidth = ((float(dataSize) * sizeof(float) * float(NUM_COPY_ITERATIONS)) / (ms * 1e6));
    printf("Bandwidth: %f GB/s\n", bandwidth);
    // Clean up.
    ::hipFree(dData);
    delete[] hData;
    ::hipEventDestroy(start);
    ::hipEventDestroy(stop);
}

int main() {
    hipStream_t stream;
    ::hipStreamCreate(&stream);

    const size_t dataSize = 1 << 20;
    checkBandwidth(stream, dataSize);
    ::hipStreamDestroy(stream);
    return 0;
}
