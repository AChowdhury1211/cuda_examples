#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../../utils/utils.cuh"

#define N (1 << 20)  // 1M elements
#define THREADS 1024  // Threads per block
#define BLOCKS (N + THREADS - 1) / THREADS  // Number of blocks

// Naive Reduction (with bank conflicts)
__global__ void reduce_naive(float *input, float *output) {
    __shared__ float shared_mem[THREADS];

    int tid = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + tid;

    // Load global memory into shared memory
    shared_mem[tid] = (global_idx < N) ? input[global_idx] : 0.0f;
    __syncthreads();

    // Reduction in shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            shared_mem[tid] += shared_mem[tid + stride];
        }
        __syncthreads();
    }

    // Write block sum to output
    if (tid == 0) output[blockIdx.x] = shared_mem[0];
}

// Optimized Reduction (Avoiding Bank Conflicts)
__global__ void reduce_optimized(float *input, float *output) {
    __shared__ float shared_mem[THREADS];

    int tid = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory (coalesced)
    shared_mem[tid] = (global_idx < N) ? input[global_idx] : 0.0f;
    __syncthreads();

    // Reduction (Avoiding Bank Conflicts)
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            shared_mem[index] += shared_mem[index + stride];
        }
        __syncthreads();
    }

    // Write block sum to output
    if (tid == 0) output[blockIdx.x] = shared_mem[0];
}

// CPU Reduction (Baseline for Benchmark)
float reduce_cpu(float *input, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        sum += input[i];
    }
    return sum;
}

// Benchmarking Function
void benchmark() {
    float *h_input, *h_output;
    float *d_input, *d_output;
    cudaCheckError(::hipHostMalloc(&h_input, N * sizeof(float)));
    cudaCheckError(::hipHostMalloc(&h_output, BLOCKS * sizeof(float)));
    cudaCheckError(::hipMalloc(&d_input, N * sizeof(float)));
    cudaCheckError(::hipMalloc(&d_output, BLOCKS * sizeof(float)));

    // Initialize input
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;  // Simple case where sum = N
    }

    cudaCheckError(::hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    cudaCheckError(::hipEventCreate((&start)));
    cudaCheckError(::hipEventCreate(&stop));
    // Benchmark Naïve Kernel
    cudaCheckError(::hipEventRecord(start));
    reduce_naive<<<BLOCKS, THREADS>>>(d_input, d_output);
    cudaCheckError(::hipEventRecord(stop));
    cudaCheckError(::hipEventSynchronize(stop));
    float time_naive;
    cudaCheckError(::hipEventElapsedTime(&time_naive, start, stop));
    printf("Naïve Kernel Time: %f ms\n", time_naive);
    cudaCheckError(::hipMemcpy(h_output, d_output, BLOCKS * sizeof(float), hipMemcpyDeviceToHost));

    // Final sum on CPU
    float sum_naive = reduce_cpu(h_output, BLOCKS);

    // Benchmark Optimized Kernel
    cudaCheckError(::hipEventRecord(start));
    reduce_optimized<<<BLOCKS, THREADS>>>(d_input, d_output);
    cudaCheckError(::hipEventRecord(stop));
    cudaCheckError(::hipEventSynchronize(stop));
    float time_optimized;
    cudaCheckError(::hipEventElapsedTime(&time_optimized, start, stop));
    printf("Optimized Kernel Time: %f ms\n", time_optimized);
    cudaCheckError(::hipMemcpy(h_output, d_output, BLOCKS * sizeof(float), hipMemcpyDeviceToHost));
    
    float sum_optimized = reduce_cpu(h_output, BLOCKS);

    // CPU Baseline
    float sum_cpu = reduce_cpu(h_input, N);

    printf("CPU Sum: %f\n", sum_cpu);
    printf("Naïve GPU Sum: %f\n", sum_naive);
    printf("Optimized GPU Sum: %f\n", sum_optimized);

    // Cleanup
    cudaCheckError(::hipHostFree(h_input));
    cudaCheckError(::hipHostFree(h_output));
    cudaCheckError(::hipFree(d_input));
    cudaCheckError(::hipFree(d_output));
}

int main() {
    benchmark();
    return 0;
}
