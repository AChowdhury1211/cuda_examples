#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include "../../utils/utils.cuh"

// Simple matmul kernel.
__global__ void matMul(float *out, const float *in1, const float *in2, const int width) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < width && col < width) {
        for (int i = 0; i < width; ++i) {
            sum += in1[row * width + i] * in2[i * width + col];
        }
        out[row * width + col] = sum;
    }
}

// Add one kernel to emphasize serial execution within a stream.
__global__ void addOne(float *out, const int width) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        int idx = row * width + col;
        out[idx] += 1.0f;
    }
}

// Minus one kernel to emphasize serial execution within a stream.
__global__ void minusOne(float *out, const int width) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        int idx = row * width + col;
        out[idx] -= 1.0f;
    }
}

// Add matrices to a final result for comparison.
__global__ void matAdd(float *out, const float *in1, const float *in2, const int width) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        out[row * width + col] = in1[row * width + col] + in2[row * width + col];
    }
}

// Perform a matmul on separate streams for a million elements, then add them.
int main() {
    // Set dimensionality.
    const int width = 1024;
    const int size = width * width * sizeof(float);

    // Create separate streams to run matmuls asynchronously.
    hipStream_t stream1, stream2;
    cudaCheckError(::hipStreamCreate(&stream1));
    cudaCheckError(::hipStreamCreate(&stream2));

    // Allocations and initializations.
    float *hA, *hB, *hC, *hD, *hResult;
    float *dA, *dB, *dC, *dD, *dResultAB, *dResultCD, *dFinal;

    // Host.
    cudaCheckError(::hipHostMalloc(&hA, size));
    cudaCheckError(::hipHostMalloc(&hB, size));
    cudaCheckError(::hipHostMalloc(&hC, size));
    cudaCheckError(::hipHostMalloc(&hD, size));
    cudaCheckError(::hipHostMalloc(&hResult, size));

    for (int i = 0; i < width * width; ++i) {
        hA[i] = 1.0f;
        hB[i] = 2.0f;
        hC[i] = 3.0f;
        hD[i] = 4.0f;
    }

    // Device.
    cudaCheckError(::hipMalloc(&dA, size));
    cudaCheckError(::hipMalloc(&dB, size));
    cudaCheckError(::hipMalloc(&dC, size));
    cudaCheckError(::hipMalloc(&dD, size));
    cudaCheckError(::hipMalloc(&dResultAB, size));
    cudaCheckError(::hipMalloc(&dResultCD, size));
    cudaCheckError(::hipMalloc(&dFinal, size));

    // Copy data using separate streams. This is not necessary, per se, but it will
    // allow for overlap of copy and execution in a batch mode case.
    cudaCheckError(::hipMemcpyAsync(dA, hA, size, hipMemcpyHostToDevice, stream1));
    cudaCheckError(::hipMemcpyAsync(dB, hB, size, hipMemcpyHostToDevice, stream1));
    cudaCheckError(::hipMemcpyAsync(dC, hC, size, hipMemcpyHostToDevice, stream2));
    cudaCheckError(::hipMemcpyAsync(dD, hD, size, hipMemcpyHostToDevice, stream2));
    
    // Launch matmul kernels on different streams.
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (width + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matMul<<<numBlocks, threadsPerBlock, 0, stream1>>>(dResultAB, dA, dB, width);
    matMul<<<numBlocks, threadsPerBlock, 0, stream2>>>(dResultCD, dC, dD, width);

    // Add 1.0f to each element in the result matrices
    addOne<<<numBlocks, threadsPerBlock, 0, stream1>>>(dResultAB, width);
    addOne<<<numBlocks, threadsPerBlock, 0, stream2>>>(dResultCD, width);

    // Subtract 1.0f to each element in the result matrices
    minusOne<<<numBlocks, threadsPerBlock, 0, stream1>>>(dResultAB, width);
    minusOne<<<numBlocks, threadsPerBlock, 0, stream2>>>(dResultCD, width);
    
    // Synchronize work prior to computing on each stream's result array.
    cudaCheckError(::hipStreamSynchronize(stream1));
    cudaCheckError(::hipStreamSynchronize(stream2));

    matAdd<<<numBlocks, threadsPerBlock, 0, stream1>>>(dFinal, dResultAB, dResultCD, width);
    cudaCheckError(::hipMemcpy(hResult, dFinal, size, hipMemcpyDeviceToHost));

    // Verify results.
    float expected = (1.0f * 2.0f + 3.0f * 4.0f) * width;
    for (int i = 0; i < width * width; ++i) {
        assert(fabs(hResult[i] - expected) < 1e-5);
    }

    std::cout << "Test passed!" << std::endl;

    // Clean up.
    cudaCheckError(::hipFree(dA));
    cudaCheckError(::hipFree(dB));
    cudaCheckError(::hipFree(dC));
    cudaCheckError(::hipFree(dD));
    cudaCheckError(::hipFree(dResultAB));
    cudaCheckError(::hipFree(dResultCD));
    cudaCheckError(::hipFree(dFinal));
    cudaCheckError(::hipStreamDestroy(stream1));
    cudaCheckError(::hipStreamDestroy(stream2));
    cudaCheckError(::hipHostFree(hA));
    cudaCheckError(::hipHostFree(hB));
    cudaCheckError(::hipHostFree(hC));
    cudaCheckError(::hipHostFree(hD));
    cudaCheckError(::hipHostFree(hResult));

    return 0;
}

