#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#include "../../utils/utils.cuh"

#define M 4096  // Rows of A, Rows of C
#define N 8192  // Columns of B, Columns of C
#define K 1024  // Columns of A, Rows of B
#define NGPUS 2 // Number of GPUs to use for computation

// Kernel to perform matrix multiplication on a portion of the matrices
__global__ void matMulKernelTP(int *A, int *B, int *C, int m, int n, int k, int col_start, int col_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int local_col = blockIdx.x * blockDim.x + threadIdx.x;
    int col = col_start + local_col;
    
    if (row < m && local_col < col_size) {
        int value = 0;
        for (int i = 0; i < k; i++) {
            value += A[row * k + i] * B[i * n + col];
        }
        //C[row * n + col] = value;  // Fixed: Use full matrix width 'n' as stride
        C[row * col_size + local_col] = value;
    }
}

// Host function for matrix multiplication
void matMulHost(const int* A, const int* B, int* C, int m, int n, int k) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int sum = 0;
            for (int p = 0; p < k; p++) {
                sum += A[i * k + p] * B[p * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

int main() {
    auto total_start = std::chrono::high_resolution_clock::now();
    // Host matrices
    int *A = new int[M * K];
    int *B = new int[K * N];
    int *C = new int[M * N]();  // GPU result
    int *C_host = new int[M * N]();  // CPU result

    // Initialize matrices with sample values
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            A[i * K + j] = (i + j) % 10;  // Simple pattern to avoid large numbers in multiplication
        }
    }
    
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            B[i * N + j] = (i + j) % 10;
        }
    }

    // Calculate split for tensor parallelism across N dimension
    const int cols_per_gpu = N / NGPUS;
    
    // Arrays for each GPU
    int *d_A[NGPUS], *d_B[NGPUS], *d_C[NGPUS];

    // Create streams and events per GPU
    hipStream_t streams[NGPUS];
    hipEvent_t startEvents[NGPUS], stopEvents[NGPUS];

    for (int gpu = 0; gpu < NGPUS; gpu++) {
        cudaCheckError(::hipSetDevice(gpu));
        cudaCheckError(::hipStreamCreate(&streams[gpu]));
        cudaCheckError(::hipEventCreate(&startEvents[gpu]));
        cudaCheckError(::hipEventCreate(&stopEvents[gpu]));
    }

    // Set up each GPU
    for (int gpu = 0; gpu < NGPUS; gpu++) {
        cudaCheckError(::hipSetDevice(gpu));
        
        // Allocate memory on current GPU
        cudaCheckError(::hipMalloc(&d_A[gpu], M * K * sizeof(int)));
        cudaCheckError(::hipMalloc(&d_B[gpu], K * N * sizeof(int)));
        cudaCheckError(::hipMalloc(&d_C[gpu], M * cols_per_gpu * sizeof(int)));
        
        // Copy input matrices to current GPU
        cudaCheckError(::hipMemcpyAsync(d_A[gpu], A, M * K * sizeof(int), hipMemcpyHostToDevice, streams[gpu]));
        cudaCheckError(::hipMemcpyAsync(d_B[gpu], B, K * N * sizeof(int), hipMemcpyHostToDevice, streams[gpu]));
        
        // Set grid and block dimensions for this GPU's portion
        const dim3 threadsPerBlock(16, 16);  // Using 16x16 thread blocks for better occupancy
        const dim3 numBlocks(
            (cols_per_gpu + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (M + threadsPerBlock.y - 1) / threadsPerBlock.y
        );
        
        const int col_start = gpu * cols_per_gpu;

        cudaCheckError(::hipEventRecord(startEvents[gpu], streams[gpu]));

        matMulKernelTP<<<numBlocks, threadsPerBlock, 0, streams[gpu]>>>(
            d_A[gpu], d_B[gpu], d_C[gpu],
            M, N, K, col_start, cols_per_gpu
        );

        // Record stop event for this GPU
        cudaCheckError(::hipEventRecord(stopEvents[gpu], streams[gpu]));
    }
    
    // Copy results back from each GPU and combine
    for (int gpu = 0; gpu < NGPUS; gpu++) {
        cudaCheckError(::hipSetDevice(gpu));
        // Copy each row's portion separately to maintain correct layout
        for (int row = 0; row < M; row++) {
            cudaCheckError(::hipMemcpyAsync(
                &C[row * N + gpu * cols_per_gpu],    // Destination in host matrix
                &d_C[gpu][row * cols_per_gpu],       // Source from GPU
                cols_per_gpu * sizeof(int),          // Size of this GPU's portion
                hipMemcpyDeviceToHost,
                streams[gpu]
            ));
        }
        // Ensure all work in stream is completed before measuring timing.
        cudaCheckError(::hipStreamSynchronize(streams[gpu]));
    }

    // Elapsed time per GPU.
    for (int gpu = 0; gpu < NGPUS; ++gpu) {
        float gpu_time = 0;
        cudaCheckError(::hipEventElapsedTime(&gpu_time, startEvents[gpu], stopEvents[gpu]));
        std::cout << "GPU " << gpu << " execution time: " << gpu_time << "ms" << std::endl;
        //printf("GPU %d execution time: %.2f ms\n", gpu, gpu_time);
    }

    for (int gpu = 0; gpu < NGPUS; gpu++) {
        cudaCheckError(::hipEventDestroy(startEvents[gpu]));
        cudaCheckError(::hipEventDestroy(stopEvents[gpu]));
        cudaCheckError(::hipStreamDestroy(streams[gpu]));
    }

    // Start CPU timing
    auto cpu_start = std::chrono::high_resolution_clock::now();
    
    // Compute result on CPU for verification
    std::cout << "Computing on CPU for verification..." << std::endl;
    matMulHost(A, B, C_host, M, N, K);
    
    // Stop CPU timing
    auto cpu_end = std::chrono::high_resolution_clock::now();
    auto cpu_duration = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end - cpu_start);
    
    // Print timing results
    std::cout << "\nTiming Results:" << std::endl;
    std::cout << "CPU Time: " << cpu_duration.count() << " ms" << std::endl;
    
    // Compare results
    bool match = true;
    int mismatch_count = 0;
    for (int i = 0; i < M * N; i++) {
        if (C[i] != C_host[i]) {
            match = false;
            mismatch_count++;
        }
    }
    
    // Output comparison results
    std::cout << "\nResult comparison:" << std::endl;
    if (match) {
        std::cout << "GPU and CPU results match perfectly!" << std::endl;
    } else {
        std::cout << "Found " << mismatch_count << " mismatches between GPU and CPU results." << std::endl;
    }
    
    // Function to print a 4x4 region of both matrices
    auto print_region = [&](const char* region_name, int start_row, int start_col) {
        std::cout << "\n" << region_name << " (4x4):" << std::endl;
        std::cout << "GPU Result:" << std::endl;
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                std::cout << C[(start_row + i) * N + (start_col + j)] << " ";
            }
            std::cout << std::endl;
        }
        
        std::cout << "CPU Result:" << std::endl;
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                std::cout << C_host[(start_row + i) * N + (start_col + j)] << " ";
            }
            std::cout << std::endl;
        }
    };
    
    // Print different regions of the matrices
    print_region("Top-left corner", 0, 0);
    print_region("Middle region", M/2 - 2, N/2 - 2);
    print_region("Bottom-right corner", M - 4, N - 4);
    
    // Free device memory on each GPU
    for (int gpu = 0; gpu < NGPUS; gpu++) {
        cudaCheckError(::hipSetDevice(gpu));
        cudaCheckError(::hipFree(d_A[gpu]));
        cudaCheckError(::hipFree(d_B[gpu]));
        cudaCheckError(::hipFree(d_C[gpu]));
    }

    // Free host memory
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_host;
    
    auto total_end = std::chrono::high_resolution_clock::now();
    auto total_duration = std::chrono::duration_cast<std::chrono::milliseconds>(total_end - total_start);
    
    std::cout << "Total Time: " << cpu_duration.count() << " ms" << std::endl;
    return 0;
}
